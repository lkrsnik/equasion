
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//example:
//k = 32
//input = 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 200, 201, 202, 203, 204, 205, 206, 207, 208, 209, 210, 211, 212, 213, 214, 215, 216, 401
int recursion(int* inputs, int current_val, int curr_index, int* result) {
	int difference = current_val - inputs[curr_index];
	if (difference == 0) {
		result[curr_index] = 1;
		return 1;
	}
	int i;
	for (i = curr_index - 1; i >= 0; i--)
		if (difference - inputs[i] >= 0) {
			if (recursion(inputs, difference, i, result)) {
				result[curr_index] = 1;
				return 1;
			}
		};
	return 0;
}

int main()
{
	int *inputs;
	int k, s, i;

	printf("Define k for equasion (there are going to be k + 1 inputs):\n");
	scanf("%d", &k);

	inputs = (int *)malloc(k * sizeof(int));

	printf("Define input of size %d:\n", k+1);
	i = 0;
	while (i < k && (scanf("%d,", &inputs[i]) == 1)) {
		i++;
	}
		
	// get final sum
	scanf("%d", &s);

	int* res = (int*)malloc(k * sizeof(int));
	
	for (i = 0; i < k; i++)
		res[i] = 0;

	// begin new recursions until result is obtained starting from highest element towards beggining
	for (i = k - 1; i >= 0; i--) 
		if(recursion(inputs, s, i, res))
			break;


	// print result
	for (i = 0; i < k; i++)
		printf("%d", res[i]);

	printf(" (");
	for (i = 0; i < k - 1; i++)
		printf("%d*%d + ", res[i], inputs[i]);
	printf("%d*%d = %d", res[k - 1], inputs[k - 1], s);
	printf(")\n");
	
	free(inputs);
	free(res);



	return 0;
}