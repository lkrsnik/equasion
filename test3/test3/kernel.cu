
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


int recursion(int* inputs, int current_val, int curr_index, int* result) {
	if (current_val - inputs[curr_index] == 0) {
		result[curr_index] = 1;
		return 1;
	}
	int i;
	for (i = curr_index - 1; i >= 0; i--) {
		if (current_val - inputs[curr_index] - inputs[i] >= 0) {
			if (recursion(inputs, current_val - inputs[curr_index], i, result)) {
				result[curr_index] = 1;
				return 1;
			}
		}
	}
	return 0;
}

int main()
{
	int i = 0;
	///*
	int *inputs;
	int n;
	int s;

	printf("Define n for equasion (there are going to be 2^n + 1 inputs):\n");
	scanf("%d", &n);

	int k = (int)pow(2, n);
	inputs = (int *)malloc(k * sizeof(int));

	printf("Define input of size %d:\n", k+1);
	while (i < k && (scanf("%d,", &inputs[i]) == 1)) {
		i++;
	}
		

	scanf("%d", &s);
	//printf("%d\n", s);
	//printf("%d", input_size);
	//for (i = 0; i < k - 1; i++)
	//	printf("%d, ", inputs[i]);
	//printf("%d\n", inputs[k - 1]);
	//*/

	/*
	int inputs[4] = { 2, 5, 14, 28 };
	int k = 4;
	int s = 30;
	*/


	int* res = (int*)malloc(k * sizeof(int));
	
	for (i = 0; i < k; i++)
		res[i] = 0;

	for (i = k - 1; i >= 0; i--) 
		if(recursion(inputs, s, i, res))
			break;

	for (i = 0; i < k; i++)
		printf("%d", res[i]);

	printf(" (");
	for (i = 0; i < k - 1; i++)
		printf("%d*%d + ", res[i], inputs[i]);
	printf("%d*%d = %d", res[k - 1], inputs[k - 1], s);
	printf(")\n");
	

	return 0;
}



/*
cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = cudaDeviceReset();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    cudaError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = cudaSetDevice(0);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = cudaGetLastError();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = cudaDeviceSynchronize();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

Error:
    cudaFree(dev_c);
    cudaFree(dev_a);
    cudaFree(dev_b);
    
    return cudaStatus;
}
*/